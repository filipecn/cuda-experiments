#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hermes/hermes.h>
#include <iostream>

using namespace hermes::cuda;

#define NX 256
#define NY 128

__global__ void NormalizeIFFT(float *g_data, int width, int height, float N) {

  // index = x * height + y

  unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

  unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

  int index = yIndex * width + xIndex;

  g_data[index] = g_data[index] / N;
}

int main() {
  hipfftReal input[NX][NY];
  for (int x = 0; x < NX; x++)
    for (int y = 0; y < NY; y++)
      input[x][y] = x * y;
  hipfftReal *d_input;
  CUDA_CHECK(hipMalloc((void **)&d_input, sizeof(hipfftReal) * NX * NY));
  CUDA_CHECK(hipMemcpy(d_input, input, sizeof(hipfftReal) * NX * NY,
                        hipMemcpyHostToDevice));
  hipfftComplex *d_output;
  CUDA_CHECK(
      hipMalloc((void **)&d_output, sizeof(hipfftComplex) * NX * (NY / 2 + 1)));

  hipfftHandle forwardPlan, inversePlan;
  if (hipfftPlan2d(&forwardPlan, NX, NY, HIPFFT_R2C) != HIPFFT_SUCCESS) {
    std::cerr << "CUFFT Error: Failed to create plan\n";
    return -1;
  }
  if (hipfftPlan2d(&inversePlan, NX, NY, HIPFFT_C2R) != HIPFFT_SUCCESS) {
    std::cerr << "CUFFT Error: Failed to create plan\n";
    return -1;
  }

  if (hipfftExecR2C(forwardPlan, d_input, d_output) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
    return -1;
  }
  CUDA_CHECK(hipDeviceSynchronize());
  if (hipfftExecC2R(inversePlan, d_output, d_input) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
    return -1;
  }
  CUDA_CHECK(hipDeviceSynchronize());

  dim3 grid(NX / 16, NY / 16, 1);
  dim3 threads(16, 16, 1);
  NormalizeIFFT<<<grid, threads>>>(d_input, NX, NY, NX * NY);

  CUDA_CHECK(hipMemcpy(input, d_input, sizeof(hipfftReal) * NX * NY,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());

  for (int x = 0; x < NX; x++)
    for (int y = 0; y < NY; y++)
      std::cerr << input[x][y] << " == " << x * y << std::endl;

  hipfftComplex output[NX * (NY / 2 + 1)];
  CUDA_CHECK(hipMemcpy(output, d_output,
                        sizeof(hipfftComplex) * NX * (NY / 2 + 1),
                        hipMemcpyDeviceToHost));

  //   for (int x = 0; x < NX; x++)
  //     std::cerr << output[x].x << " " << output[x].y << std::endl;

  hipfftDestroy(forwardPlan);
  hipfftDestroy(inversePlan);
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}