#include "hip/hip_runtime.h"
#include "io.h"

using namespace heliades;

__global__ void __setupScene(hitable **objects, hitable **world, camera **cam) {
  using namespace hermes::cuda;
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    objects[0] = new sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f,
                            new lambertian(vec3(0.1, 0.2, 0.5)));
    objects[1] = new sphere(vec3(0.0f, -100.5f, -1.0f), 100.f,
                            new lambertian(vec3(0.8, 0.8, 0.0)));
    objects[2] = new sphere(vec3(1.0f, 0.0f, -1.0f), 0.5f,
                            new metal(vec3(0.8, 0.6, 0.2)));
    objects[3] =
        new sphere(vec3(-1.0f, 0.0f, -1.0f), 0.5f, new dielectric(1.5));
    objects[4] =
        new sphere(vec3(-1.0f, 0.0f, -1.0f), -0.45f, new dielectric(1.5));
    *cam = new camera();
    *world = new hitableList(objects, 5);
  }
}

__device__ hermes::cuda::vec3 color(const heliades::ray &r, hitable **world,
                                    HS &rng, int level = 0) {
  using namespace hermes::cuda;
  hitRecord rec;
  if (level < 5 &&
      (*world)->hit(r, 0.00001, hermes::cuda::Constants::greatest<float>(),
                    rec)) {
    ray scattered;
    vec3 attenuation;
    if (level < 5 &&
        rec.materialPtr->scatter(r, rec, attenuation, scattered, rng))
      return attenuation * color(scattered, world, rng, level + 1);
    return vec3(0, 0, 0);
  }
  vec3 d = hermes::cuda::normalize(r.d);
  float t = 0.5 * (d.y + 1.0);
  return vec3(1.0f - t) + t * vec3(0.5, 0.7, 1.0);
}

__global__ void __render(heliades::Film::Pixel *out, int w, int h,
                         hitable **world, camera **cam, int ns) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int index = y * w + x;
  // printf("%d\n", index);
  if (x < w && y < h) {
    HS rng, sng;
    rng.setIndex(index + 1);
    sng.setIndex(index + 1);
    hermes::cuda::vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
      auto sd = sng.random2();
      float u = float(x + sd.x) / float(w);
      float v = float(y + sd.y) / float(h);
      heliades::ray r = (*cam)->getRay(u, v);
      col += color(r, world, rng);
    }
    col /= float(ns);
    col = hermes::cuda::vec3(sqrt(col.r()), sqrt(col.g()), sqrt(col.b()));
    out[index].xyz[0] = 255.99f * col.r();
    out[index].xyz[1] = 255.99f * col.g();
    out[index].xyz[2] = 255.99f * col.b();
  }
}

int main(int argc, char **argv) {
  // scene
  hitable **objects = nullptr;
  hitable **world = nullptr;
  camera **cam = nullptr;
  {
    using namespace hermes::cuda;
    CUDA_CHECK(hipMalloc(&objects, 5 * sizeof(hitable *)));
    CUDA_CHECK(hipMalloc(&world, sizeof(hitable *)));
    CUDA_CHECK(hipMalloc(&cam, sizeof(camera *)));
  }
  __setupScene<<<1, 1>>>(objects, world, cam);
  // rendering
  hermes::cuda::vec2u imageSize(800, 400);
  heliades::Film film(imageSize);
  hermes::ThreadArrayDistributionInfo td(imageSize.x, imageSize.y);
  __render<<<td.gridSize, td.blockSize>>>(film.pixelsDeviceData(), imageSize.x,
                                          imageSize.y, world, cam, 50);
  std::cerr << "render complete\n";
  // hipFree(scene.list[0]);
  // hipFree(scene.list[1]);
  // VIS
  circe::SceneApp<> app(imageSize.x, imageSize.y, "", false);
  app.addViewport2D(0, 0, imageSize.x, imageSize.y);
  CudaOpenGLInterop cgl(imageSize.x, imageSize.y);
  pixelsToTexture(imageSize.x, imageSize.y, film.pixelsDeviceData(),
                  cgl.bufferPointer());
  cgl.sendToTexture();
  circe::ScreenQuad screen;
  app.renderCallback = [&]() {
    screen.shader->begin();
    screen.shader->setUniform("tex", 0);
    cgl.bindTexture(GL_TEXTURE0);
    screen.render();
  };
  app.run();
  return 0;
}